
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#ifdef _MSC_VER
# define strdup _strdup
#endif

__global__ void kernel()
{
    printf("hello from GPU\n");
}

struct Error {
    int code;
    char *message;
};

extern "C"
#ifdef _WIN32
__declspec(dllexport)
#else
__attribute__((visibility("default")))
#endif
Error cuda_func(void *ptr)
{
    kernel<<<1,1>>>();
    auto err = hipGetLastError();
    if (err != hipSuccess)
        return {err, strdup(hipGetErrorString(err))};
    err = hipDeviceSynchronize();
    return {err, strdup(hipGetErrorString(err))};
    (void)ptr;
}
